#include <stdio.h>
#include <hip/hip_runtime.h>

//-----------------------------------------------------------------------------
// TheKernel: basic kernel containing a print statement.
//-----------------------------------------------------------------------------
__global__ void TheKernel()
{
  // Give the kernel something to keep its (single) thread occupied
  int i, j, k;
  k = 0;
  for (i = 0; i < 1000; i++) {
    for (j = 0; j < 1000; j++) {
      k += i;
      if (k > 2000) {
	k -= 4*j;
      }
      else {
	k += j;
      }
    }
  }
  
  printf("This is the kernel saying hello world, from the GPU.\n");
}

//-----------------------------------------------------------------------------
// main
//-----------------------------------------------------------------------------
int main()
{
  printf("This is the C layer saying hello world, from the host.\n");
  TheKernel<<<1, 1>>>();

  // Device synchronization
  hipDeviceSynchronize();
  printf("LOOK: device synchronization stops the host until the kernel is "
	 "done.\n");
  
  // It appears essential to call for synchronization before finally
  // exiting, lest you risk the program crashing your machine!
  hipDeviceSynchronize();

  return 0;
}
