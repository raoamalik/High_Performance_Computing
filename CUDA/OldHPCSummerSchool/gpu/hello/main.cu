/* 
 *
 *  hello world ... from a GPU!
 *
 */

#include <sstream>
#include <cassert>

#include<stdio.h>

#include<hip/hip_runtime.h>


__global__ void hello() {
    printf("thread %5i from block %5i says, \"hello, world!\"\n",threadIdx.x,blockIdx.x);
}

// main!
int main (int argc, char* argv[]) {

    if ( argc != 3 ) {
        printf("\n");
        printf("    hello.x -- hello from a gpu!\n");
        printf("\n");
        printf("    usage: ./hello.x n nblocks nthreads_per_block\n");
        printf("\n");
        printf("    nblocks:            number of blocks\n");
        printf("    nthreads_per_block: number of threads per block\n");
        printf("\n");
        exit(EXIT_FAILURE);
    }
    printf("\n");

    std::stringstream ss; ss << argv[1] << " " << argv[2];
    int nblocks; ss >> nblocks;
    int nthreads_per_block; ss >> nthreads_per_block;

    hello<<<nblocks,nthreads_per_block>>>();

    // check for errors
    hipError_t error = hipGetLastError();
    if (error!=hipSuccess) {
       printf("\n");
       printf("    error: %s\n\n", hipGetErrorString(error) );
       printf("\n");
       exit(EXIT_FAILURE);
    }

    hipDeviceReset();

    printf("\n");

    return 0;
}
